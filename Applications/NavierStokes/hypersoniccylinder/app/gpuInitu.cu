
#include <hip/hip_runtime.h>
template <typename T>  __global__  void kernelgpuInitu(T *f, T *xdg, T *uinf, T *param, int ng, int ncx, int nce, int npe, int ne)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		int j = i%npe;
		int k = (i-j)/npe;
		T param1 = param[0];
		T param2 = param[1];
		T param3 = param[2];
		T param4 = param[3];
		T param5 = param[4];
		T param6 = param[5];
		T param7 = param[6];
		T param8 = param[7];
		T param9 = param[8];
		T param10 = param[9];
		T param11 = param[10];
		T param12 = param[11];
		T param13 = param[12];
		T param14 = param[13];
		T param15 = param[14];
		T param16 = param[15];
		T param17 = param[16];
		T uinf1 = uinf[0];
		T uinf2 = uinf[1];
		T xdg1 = xdg[j+npe*0+npe*ncx*k];
		T xdg2 = xdg[j+npe*1+npe*ncx*k];
		f[j+npe*0+npe*nce*k] = param5;
		f[j+npe*1+npe*nce*k] = param6;
		f[j+npe*2+npe*nce*k] = param7;
		f[j+npe*3+npe*nce*k] = param8;
		i += blockDim.x * gridDim.x;
	}
}

template <typename T> void gpuInitu(T *f, T *xdg, T *uinf, T *param, int ng, int ncx, int nce, int npe, int ne)
{
	int blockDim = 256;
	int gridDim = (ng + blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuInitu<<<gridDim, blockDim>>>(f, xdg, uinf, param, ng, ncx, nce, npe, ne);
}

template void gpuInitu(double *, double *, double *, double *, int, int, int, int, int);
template void gpuInitu(float *, float *, float *, float *, int, int, int, int, int);
