
#include <hip/hip_runtime.h>
template <typename T>  __global__  void kernelgpuFlux(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		T param1 = param[0];
		T param2 = param[1];
		T param3 = param[2];
		T param7 = param[6];
		T udg1 = udg[0*ng+i];
		T udg2 = udg[1*ng+i];
		T udg3 = udg[2*ng+i];
		T udg4 = udg[3*ng+i];
		T udg5 = udg[4*ng+i];
		T udg6 = udg[5*ng+i];
		T udg7 = udg[6*ng+i];
		T udg8 = udg[7*ng+i];
		T udg9 = udg[8*ng+i];
		T udg10 = udg[9*ng+i];
		T udg11 = udg[10*ng+i];
		T udg12 = udg[11*ng+i];
		T t2 = udg2*udg2;
		T t3 = 1.0/(udg1*udg1);
		T t4 = 1.0/udg1;
		T t5 = t2*t3*(1.0/2.0);
		T t6 = udg3*udg3;
		T t7 = t3*t6*(1.0/2.0);
		T t8 = t5+t7;
		T t11 = t8*udg1;
		T t9 = -t11+udg4;
		T t10 = param1-1.0;
		T t12 = 1.0/param1;
		T t22 = param7*t12;
		T t13 = param7-t22;
		T t14 = 1.0/(t13*t13);
		T t15 = t4*t9*t10*t14;
		T t16 = sqrt(t15);
		T t28 = t4*udg3*udg5;
		T t17 = -t28+udg7;
		T t18 = t4*t17;
		T t31 = t4*udg2*udg9;
		T t19 = -t31+udg10;
		T t20 = t4*t19;
		T t21 = t18+t20;
		T t27 = t4*udg2*udg5;
		T t23 = -t27+udg6;
		T t24 = t4*t23*2.0;
		T t35 = t4*udg3*udg9;
		T t25 = -t35+udg11;
		T t26 = t24-t4*t25;
		T t29 = 1.0/t13;
		T t30 = t4*udg2*udg3;
		T t32 = param2*t16*t21;
		T t33 = t30+t32;
		T t34 = t9*t10;
		T t36 = t4*udg4;
		T t37 = t4*t9*t10;
		T t38 = t36+t37;
		T t39 = t4*t23;
		T t40 = t39-t4*t25*2.0;
		T t41 = t4*t9*t10*t29;
		T t42 = pow(t41,3.0/4.0);
		f[0*ng+i] = udg2;
		f[1*ng+i] = t34+t2*t4+param2*t16*t26*(2.0/3.0);
		f[2*ng+i] = t33;
		f[3*ng+i] = t38*udg2+param2*t4*t16*t21*udg3+param2*t4*t16*t26*udg2*(2.0/3.0)-param3*t3*t29*t42*(t10*udg1*(-udg8+t8*udg5+udg1*(t3*t17*udg3+t3*t23*udg2))+t9*t10*udg5);
		f[4*ng+i] = udg3;
		f[5*ng+i] = t33;
		f[6*ng+i] = t34+t4*t6-param2*t16*t40*(2.0/3.0);
		f[7*ng+i] = t38*udg3+param2*t4*t16*t21*udg2-param2*t4*t16*t40*udg3*(2.0/3.0)-param3*t3*t29*t42*(t10*udg1*(-udg12+t8*udg9+udg1*(t3*t19*udg2+t3*t25*udg3))+t9*t10*udg9);
		i += blockDim.x * gridDim.x;
	}
}

template <typename T> void gpuFlux(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int blockDim = 256;
	int gridDim = (ng + blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuFlux<<<gridDim, blockDim>>>(f, xdg, udg, odg, wdg, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
}

template void gpuFlux(double *, double *, double *, double *, double *, double *, double *, double, int, int, int, int, int, int, int, int);
template void gpuFlux(float *, float *, float *, float *, float *, float *, float *, float, int, int, int, int, int, int, int, int);
