
#include <hip/hip_runtime.h>
template <typename T>  __global__  void kernelgpuTdfunc(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		f[0*ng+i] = 1.0;
		f[1*ng+i] = 1.0;
		f[2*ng+i] = 1.0;
		f[3*ng+i] = 1.0;
		i += blockDim.x * gridDim.x;
	}
}

template <typename T> void gpuTdfunc(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int blockDim = 256;
	int gridDim = (ng + blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuTdfunc<<<gridDim, blockDim>>>(f, xdg, udg, odg, wdg, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
}

template void gpuTdfunc(double *, double *, double *, double *, double *, double *, double *, double, int, int, int, int, int, int, int, int);
template void gpuTdfunc(float *, float *, float *, float *, float *, float *, float *, float, int, int, int, int, int, int, int, int);
